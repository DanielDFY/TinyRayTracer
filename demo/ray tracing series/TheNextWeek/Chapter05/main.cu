#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <ctime>
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

#include <hip/hip_runtime.h>
#include <>

#include <Color.cuh>
#include <Ray.cuh>
#include <Camera.cuh>
#include <Sphere.cuh>
#include <HittableList.cuh>
#include <Material.cuh>

#include <helperUtils.cuh>
#include <hiprand/hiprand_kernel.h>

using namespace TinyRT;

constexpr int objNum = 1;
constexpr int imageTexNum = 1;

struct TextureData {
	hipTextureObject_t textureObject;
	int width, height;
	hipArray* deviceData;
};

TextureData loadAndInitTexture(const char* fileName) {
	int width, height, depth;
	const auto texData = stbi_load(fileName, &width, &height, &depth, 0);
	const auto pixelNum = width * height;
	const auto imageSize = pixelNum * depth;
	float* hostData = new float[imageSize];
	for (unsigned int layer = 0; layer < 3; layer++)
		for (auto i = 0; i < pixelNum; i++)
			hostData[layer * pixelNum + i] = texData[i * 3 + layer] / 255.0f;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipArray* deviceData;
	hipMalloc3DArray(&deviceData, &channelDesc, make_hipExtent(width, height, 3), hipArrayLayered);

	hipMemcpy3DParms memcpy3DParms = { 0 };
	memcpy3DParms.srcPos = make_hipPos(0, 0, 0);
	memcpy3DParms.dstPos = make_hipPos(0, 0, 0);
	memcpy3DParms.srcPtr = make_hipPitchedPtr(hostData, width * sizeof(float), width, height);
	memcpy3DParms.dstArray = deviceData;
	memcpy3DParms.extent = make_hipExtent(width, height, 3);
	memcpy3DParms.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&memcpy3DParms);

	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = deviceData;
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(hipTextureDesc));
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = true;

	hipTextureObject_t textureObject;
	hipCreateTextureObject(&textureObject, &texRes, &texDesc, nullptr);

	return { textureObject, width, height, deviceData };
}

void cleanTexture(TextureData textureData) {
	hipDestroyTextureObject(textureData.textureObject);
	hipFreeArray(textureData.deviceData);
}

__device__ void earth(Hittable** hittablePtrList, hiprandState* const objRandStatePtr, Texture** texturePtrList, TextureData* textureDataList) {
	size_t objIdx = 0;
	size_t textureIdx = 0;

	TextureData earthTextureData = textureDataList[textureIdx];
	texturePtrList[textureIdx] = new ImageTexture(earthTextureData.textureObject, earthTextureData.width, earthTextureData.height);
	hittablePtrList[objIdx] = new Sphere(Point3(0, 0, 0), 2, new Lambertian(texturePtrList[textureIdx]));
}

__device__ Color rayColor(const Ray& r, Hittable** hittablePtr, const int maxDepth, hiprandState* const randStatePtr) {
	Ray curRay = r;
	Vec3 curAttenuation(1.0f, 1.0f, 1.0f);
	for (size_t i = 0; i < maxDepth; ++i) {
		HitRecord rec;
		if ((*hittablePtr)->hit(curRay, 0.001f, M_FLOAT_INFINITY, rec)) {
			Ray scattered;
			Vec3 attenuation;
			if (rec.matPtr->scatter(curRay, rec, attenuation, scattered, randStatePtr)) {
				curRay = scattered;
				curAttenuation *= attenuation;
			} else {
				return { 0.0f, 0.0f, 0.0f };
			}
		} else {
			const Vec3 unitDirection = unitVec3(curRay.direction());
			const double t = 0.5f * (unitDirection.y() + 1.0f);
			const Color background = (1.0f - t) * Color(1.0f, 1.0f, 1.0f) + t * Color(0.5f, 0.7f, 1.0f);
			return curAttenuation * background;
		}
	}
	// exceed max depth
	return { 0.0f, 0.0f, 0.0f };
}

__global__ void renderInit(const int imageWidth, const int imageHeight, hiprandState* const randStateList, unsigned int seed) {
	const int col = threadIdx.x + blockIdx.x * blockDim.x;
	const int row = threadIdx.y + blockIdx.y * blockDim.y;
	if ((col >= imageWidth) || (row >= imageHeight))
		return;

	const int idx = row * imageWidth + col;

	// init random numbers for anti-aliasing
	// each thread gets its own special seed, fixed sequence number, fixed offset
	hiprand_init(seed + idx, 0, 0, &randStateList[idx]);
}

__global__ void render(
	Color* const pixelBuffer,
	const int imageWidth,
	const int imageHeight,
	Camera** const camera,
	hiprandState* const pixelRandStateList,
	const int samplesPerPixel,
	const int maxDepth,
	Hittable** const hittablePtrList) {

	const int col = threadIdx.x + blockIdx.x * blockDim.x;
	const int row = threadIdx.y + blockIdx.y * blockDim.y;
	if (col >= imageWidth || row >= imageHeight)
		return;

	const int idx = row * imageWidth + col;

	hiprandState randState = pixelRandStateList[idx];
	Color pixelColor(0.0f, 0.0f, 0.0f);
	for (size_t s = 0; s < samplesPerPixel; ++s) {
		const auto u = (static_cast<float>(col) + randomFloat(&randState)) / static_cast<float>(imageWidth - 1);
		const auto v = 1.0 - (static_cast<float>(row) + randomFloat(&randState)) / static_cast<float>(imageHeight - 1);

		const Ray r = (*camera)->getRay(u, v, &randState);

		pixelColor += rayColor(r, hittablePtrList, maxDepth, &randState);
	}

	pixelColor /= samplesPerPixel;
	pixelColor.gammaCorrect();

	pixelBuffer[idx] = pixelColor;
}

__global__ void createInit(hiprandState* const randStatePtr, unsigned int seed) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// init a random number for sphere generating
		hiprand_init(seed, 0, 0, randStatePtr);
	}
}

__global__ void createWorld(Camera** camera, float aspectRatio, Hittable** hittablePtrList, Hittable** hittableWorldObjListPtr, hiprandState* objRandStatePtr, Texture** texturePtrList, TextureData* textureDataList) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		const Point3 lookFrom(13.0f, 2.0f, -3.0f);
		const Point3 lookAt(0.0f, 0.0f, 0.0f);
		const Vec3 vUp(0.0f, 1.0f, 0.0f);
		const float vFov = 20.0f;
		const float aperture = 0.0f;
		const float distToFocus = 10.0f;
		const float time0 = 0.0f;
		const float time1 = 1.0f;
		
		*camera = new Camera(lookFrom, lookAt, vUp, vFov, aspectRatio, aperture, distToFocus, time0, time1);

		earth(hittablePtrList, objRandStatePtr, texturePtrList, textureDataList);

		*hittableWorldObjListPtr = new HittableList(hittablePtrList, objNum);
	}
}

__global__ void freeWorld(Camera** camera, Hittable** hittableList, size_t hittableNum, Hittable** hittableWorldObjList, Texture** texturePtrList) {
	delete* camera;
	for (int i = 0; i < hittableNum; ++i) {
		// delete material instances
		delete hittableList[i]->matPtr();
		// delete object instances
		delete hittableList[i];
	}

	for (int i = 0; i < imageTexNum; ++i) {
		// delete texture instances
		delete texturePtrList[i];
	}
	
	delete* hittableWorldObjList;
}

int main() {
	/* image config */
	constexpr float aspectRatio = 16.0f / 9.0f;
	constexpr int imageWidth = 800;
	constexpr int imageHeight = static_cast<int>(imageWidth / aspectRatio);
	constexpr int samplesPerPixel = 20;
	constexpr int maxDepth = 5;

	/* image output file */
	const std::string fileName("output.png");

	/* thread block config */
	constexpr int threadBlockWidth = 16;
	constexpr int threadBlockHeight = 16;

	// preparation
	constexpr int channelNum = 3; // rgb
	constexpr int pixelNum = imageWidth * imageHeight;

	// allocate memory for pixel buffer
	const auto pixelBufferPtr = cudaManagedUniquePtr<Color>(pixelNum * sizeof(Color));

	// allocate random state
	const auto seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
	const auto objRandStatePtr = cudaUniquePtr<hiprandState>(sizeof(hiprandState));
	const auto pixelRandStateListPtr = cudaUniquePtr<hiprandState>(pixelNum * sizeof(hiprandState));

	// create world of hittable objects and the camera
	const auto cameraPtr = cudaUniquePtr<Camera*>(sizeof(Camera*));
	const auto hittablePtrList = cudaUniquePtr<Hittable*>(objNum * sizeof(Hittable*));
	const auto hittableWorldObjListPtr = cudaUniquePtr<Hittable*>(sizeof(Hittable*));

	const auto texturePtrList = cudaUniquePtr<Texture*>(imageTexNum * sizeof(Texture*));
	const auto textureDataList = cudaManagedUniquePtr<TextureData>(imageTexNum * sizeof(TextureData));

	// load and initiate earth texture
	textureDataList.get()[0] = loadAndInitTexture("earthmap.jpg");
	
	createInit<<<1, 1>>>(objRandStatePtr.get(), seed);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	createWorld<<<1, 1>>>(cameraPtr.get(), aspectRatio, hittablePtrList.get(), hittableWorldObjListPtr.get(), objRandStatePtr.get(), texturePtrList.get(), textureDataList.get());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// start timer
	const clock_t start = clock();

	const dim3 blockDim(imageWidth / threadBlockWidth + 1, imageHeight / threadBlockHeight + 1);
	const dim3 threadDim(threadBlockWidth, threadBlockHeight);

	// render init
	renderInit<<<blockDim, threadDim>>>(imageWidth, imageHeight, pixelRandStateListPtr.get(), seed);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// render the image into buffer
	render<<<blockDim, threadDim>>>(
		pixelBufferPtr.get(),
		imageWidth,
		imageHeight,
		cameraPtr.get(),
		pixelRandStateListPtr.get(),
		samplesPerPixel,
		maxDepth,
		hittableWorldObjListPtr.get()
	);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// stop timer
	const clock_t stop = clock();

	// measure rendering time
	const auto renderingMillisecond = stop - start;

	// other image writer arguments
	constexpr int imageSize = pixelNum * channelNum;
	constexpr size_t strideBytes = imageWidth * channelNum * sizeof(unsigned char);
	const std::unique_ptr<unsigned char[]> pixelDataPtr(new unsigned char[imageSize]);

	// store the pixel data into writing buffer as 8bit color
	for (int pixelIdx = 0, dataIdx = 0; pixelIdx < pixelNum; ++pixelIdx) {
		const Color color = pixelBufferPtr.get()[pixelIdx];
		pixelDataPtr[dataIdx++] = static_cast<unsigned char>(color.r8bit());
		pixelDataPtr[dataIdx++] = static_cast<unsigned char>(color.g8bit());
		pixelDataPtr[dataIdx++] = static_cast<unsigned char>(color.b8bit());
	}

	// print rendering time
	std::cout << "Complete!\n" << "The rendering took " << renderingMillisecond << "ms" << std::endl;

	// write pixel data to output file
	stbi_write_png(fileName.c_str(), imageWidth, imageHeight, channelNum, pixelDataPtr.get(), strideBytes);

	// free resources
	freeWorld<<<1, 1>>>(cameraPtr.get(), hittablePtrList.get(), objNum, hittableWorldObjListPtr.get(), texturePtrList.get());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
	//cleanTexture(earthTextureData);
	
	return 0;
}